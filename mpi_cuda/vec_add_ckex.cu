#include "hip/hip_runtime.h"
﻿
/*  Bill Derksen - 2/14/2021

    Example / test CUDA program. Sums two vectors.

*/

//cuda
#include "hip/hip_runtime.h"
#include ""

//std
#include <stdio.h>
#include <iostream>
#include <stdexcept>
#include <cassert>
#include <chrono>
#include <iomanip>

void printDeviceWarpSize();

// KERNEL
__global__ void sampleAddKernel(float *a, float* b, float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

// MAIN
int main()
{
    std::cout << "Starting cuda program...\n";
    printDeviceWarpSize();

    float * host_data_a,* host_data_b, *host_data_c;
    float * device_data_a,* device_data_b, *device_data_c;

    int N, nBytes;

    N = 1024;// n elements
    nBytes = N * sizeof(float);

    // Allocate host (CPU) memory
    host_data_a = (float*)malloc(nBytes);
    host_data_b = (float*)malloc(nBytes);
    host_data_c = (float*)malloc(nBytes);

    // Allocate device (GPU) memory
    try {
        hipError_t gpuMalloc1 = hipMalloc((void**)&device_data_a, nBytes);
        hipError_t gpuMalloc2 = hipMalloc((void**)&device_data_b, nBytes);
        hipError_t gpuMalloc3 = hipMalloc((void**)&device_data_c, nBytes);

        if (gpuMalloc1 != hipSuccess || gpuMalloc2 != hipSuccess || gpuMalloc3 != hipSuccess)
            throw std::runtime_error("Failed to allocate device memory!!!");
    }
    catch (std::exception e) {
        std::cerr << e.what();
        return -1;
    }

    // Populate data arrays
    for (int i = 0; i < N; i++) {
        host_data_a[i] =  i;
        host_data_b[i] = (i*i);
    }

    /* START TIME */
    auto start = std::chrono::high_resolution_clock::now();

    // Copy data from host-->device (cpu to gpu)
    try {
        hipError_t memCpy1 = hipMemcpy(device_data_a, host_data_a, nBytes, hipMemcpyHostToDevice);
        hipError_t memCpy2 = hipMemcpy(device_data_b, host_data_b, nBytes, hipMemcpyHostToDevice);

        if (memCpy1 != hipSuccess || memCpy2 != hipSuccess)
            throw std::runtime_error("Failed to copy host data to device!!!");
    }
    catch (std::exception e) {
        std::cerr << e.what();
        return -1;
    }
    
    // Launch kernel
    // Note: N threads per block should be multiple of 32 (= 1 warp)
    sampleAddKernel<<<1, N>>>(device_data_a, device_data_b, device_data_c);

    // Copy result back to device memory
    try {
        hipError_t memCpy1 = hipMemcpy(host_data_c, device_data_c, nBytes, hipMemcpyDeviceToHost);

        if (memCpy1 != hipSuccess)
            throw std::runtime_error("Failed to copy device data to host!!!");
    }
    catch (std::exception e) {
        std::cerr << e.what();
        return -1;
    }

    /* END TIME */
    auto end = std::chrono::high_resolution_clock::now();
    double time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
    time_taken *= 1e-6;
    std::cout << "Execution time: " << std::setprecision(9) << time_taken << " ms\n";

    // Verify Results: should be i + i^2
    for (int i = 0; i < N; i++) {
        assert(host_data_c[i] == (host_data_a[i] + host_data_b[i]), "Result is incorrect!!!");
    }

    std::cout << "Vector addition complete and correct!!!\n";

    // Free memory
    free(host_data_a);
    free(host_data_b);
    free(host_data_c);
    hipFree(device_data_a);
    hipFree(device_data_b);
    hipFree(device_data_c);

    std::cout << "Program finished, exiting...\n";
    return 0;
}

// Fxn that gets and prints device's warp size. Typically is 32, but still good to check...
void printDeviceWarpSize() {

    hipDeviceProp_t deviceProperties;
    if (hipGetDeviceProperties(&deviceProperties, 0) != hipSuccess)
        std::cout << "Failed to retrieve device properties!!!\n";
    else
        std::cout << "Current device's warp size is: " << deviceProperties.warpSize << '\n';
}